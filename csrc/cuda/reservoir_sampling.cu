#include "hip/hip_runtime.h"
#include "reservoir_sampling.cuh"

int const threadsPerBlock = 512;

__global__ void generate_samples(
  int64_t *samples,
  int k,
  hiprandStateMtgp32 *state
){
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  samples[thread_id] = hiprand(state) % (thread_id + k + 1);
}

__global__ void generate_reservoir(
  int64_t *indices,
  int64_t *samples,
  int nb_iterations,
  int k
){
  for(int i = 0; i < nb_iterations; i++){
    int64_t z = samples[i];
    if (z < k) {
      thrust::swap(indices[z], indices[i + k]);
    }
  }
}

torch::Tensor reservoir_sampling_cuda(torch::Tensor& x, int k){

  if (!x.is_contiguous()){
    x = x.contiguous();
  }

  int n = x.numel();
  auto options = x.options().dtype(torch::kLong);
  torch::Tensor indices_n = torch::arange({n}, options);

  THCState *state = at::globalContext().lazyInitCUDA();
  THCRandom_seed(state);
  THCGenerator *generator = THCRandom_getGenerator(state);

  int split, begin, end;

  if(2 * k < n){
    split = n - k;
    begin = n - k;
    end = n;
  } else {
    split = k;
    begin = 0;
    end = k;
  }

  int nb_iterations = std::min(k, n - k);
  dim3 blocks((nb_iterations + threadsPerBlock - 1)/threadsPerBlock);
  dim3 threads(threadsPerBlock);

  torch::Tensor samples = torch::arange({nb_iterations}, options);

  generate_samples<<<blocks, threads>>>(
    samples.data<int64_t>(),
    split,
    generator->state.gen_states
  );

  generate_reservoir<<<1, 1>>>(
    indices_n.data<int64_t>(),
    samples.data<int64_t>(),
    nb_iterations,
    split
  );

  return x.index_select(
    0,
    indices_n.index_select(
      0,
      torch::arange(begin, end, options)
    )
  );

}
